#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include "maskRCNNKernels.h"
#include "plugin.h"
#include <NvInfer.h>
#include <assert.h>
#include <iostream>
#include <stdio.h>
// #include <thrust/device_ptr.h>
// #include <thrust/fill.h>

#define DUBUG_KERNEL 0
#define DUBUG_BATCH 0
#define DEBUG_T 1

#define dMIN(a, b) ((a) < (b) ? (a) : (b))
#define dMAX(a, b) ((a) > (b) ? (a) : (b))
#define dCLAMP(x, xMin, xMax) ((x) > (xMin) ? ((x) < (xMax) ? (x) : (xMax)) : (xMin))



__global__ void resize_nearest_kernel_2d(int nbatch, float scale, int2 osize, float const* idata, int istride,
    int ibatchstride, float* odata, int ostride, int obatchstride)
{

    int x0 = threadIdx.x + blockIdx.x * blockDim.x;
    int y0 = threadIdx.y + blockIdx.y * blockDim.y;
    int z0 = blockIdx.z;
    for (int batch = z0; batch < nbatch; batch += gridDim.z)
    {
        for (int oy = y0; oy < osize.y; oy += blockDim.y * gridDim.y)
        {
            for (int ox = x0; ox < osize.x; ox += blockDim.x * gridDim.x)
            {
                int ix = int(ox / scale);
                int iy = int(oy / scale);
                odata[batch * obatchstride + oy * ostride + ox] = idata[batch * ibatchstride + iy * istride + ix];
            }
        }
    }
}

void resizeNearest(dim3 grid, dim3 block, hipStream_t stream, int nbatch, float scale, int2 osize, float const* idata,
    int istride, int ibatchstride, float* odata, int ostride, int obatchstride)
{

    resize_nearest_kernel_2d<<<grid, block, 0, stream>>>(
        nbatch, scale, osize, idata, istride, ibatchstride, odata, ostride, obatchstride);
}


